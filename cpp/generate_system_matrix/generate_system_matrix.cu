#include "hip/hip_runtime.h"
#define GOOGLE_CUDA 1
#if GOOGLE_CUDA
#define EIGEN_USE_GPU
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#define abs(x) ((x)<0 ? (-x) : (x))

const int GRIDDIM = 32;
const int BLOCKDIM = 1024;

__device__ int project_device(const float x1_, const float y1_, const float z1_,
                               const float x2_, const float y2_, const float z2_,
                               const int nx, const int ny, const int nz,
                               const float cx, const float cy, const float cz,
                               const float sx, const float sy, const float sz,
                               const float *image)
{
    const float dx_ = sx / nx;
    const float dx = 1.0f;
    const float dy = sy / ny / dx_;
    const float dz = sz / nz / dx_;
    const float x1 = (x1_ - cx) / dx_;
    const float x2 = (x2_ - cx) / dx_;
    const float y1 = (y1_ - cy) / dx_;
    const float y2 = (y2_ - cy) / dx_;
    const float z1 = (z1_ - cz) / dx_;
    const float z2 = (z2_ - cz) / dx_;

    const float xd = x2 - x1;
    const float yd = y2 - y1;
    const float zd = z2 - z1;

    if (sqrt(xd * xd + yd * yd) < 10.0f) {return;}

    const float nx2 = nx / 2.0f;
    const float ny2 = ny / 2.0f;
    const float nz2 = nz / 2.0f;

    const float L = sqrt(xd * xd + yd * yd + zd * zd);
    vproj[0] = 0.0f;

    if (abs(xd) > abs(yd))
    {
        float ky = yd / xd;
        float kz = zd / xd;

        for (int ix = 0; ix < nx; ++ix)
        {
            float xx1 = ix - nx2;
            float xx2 = xx1 + 1.0f;
            float yy1, yy2, zz1, zz2;

            if (ky >= 0.0f)
            {
                yy1 = (y1 + ky * (xx1 - x1)) / dy + ny2;
                yy2 = (y1 + ky * (xx2 - x1)) / dy + ny2;

            }
            else
            {
                yy1 = (y1 + ky * (xx2 - x1)) / dy + ny2;
                yy2 = (y1 + ky * (xx1 - x1)) / dy + ny2;
            }
            int cy1 = (int)floor(yy1);
            int cy2 = (int)floor(yy2);

            if (kz >= 0.0f)
            {
                zz1 = (z1 + kz * (xx1 - x1)) / dz + nz2;
                zz2 = (z1 + kz * (xx2 - x1)) / dz + nz2;
            }
            else
            {
                zz1 = (z1 + kz * (xx2 - x1)) / dz + nz2;
                zz2 = (z1 + kz * (xx1 - x1)) / dz + nz2;
            }
            int cz1 = (int)floor(zz1);
            int cz2 = (int)floor(zz2);

            if (cy1 == cy2)
            {
                if (0 <= cy1 && cy1 < ny)
                {
                    if (cz1 == cz2)
                    {
                        if (0 <= cz1 && cz1 < nz)
                        {
                            float weight = sqrt(1 + ky * ky + kz * kz) * dx_ / L / L;
                            vproj[0] += image[ix + cy1 * nx + cz1 * nx * ny] * weight;
                        }
                    }
                    else
                    {
                        if (-1 <= cz1 and cz1 < nz)
                        {
                            float rz = (cz2 - zz1) / (zz2 - zz1);
                            if (cz1 >= 0)
                            {
                                float weight = rz * sqrt(1 + ky * ky + kz * kz) * dx_ / L / L;
                                vproj[0] += image[ix + cy1 * nx + cz1 * nx * ny] * weight;
                            }

                            if (cz2 < nz)
                            {
                                float weight = (1 - rz) * sqrt(1 + ky * ky + kz * kz) * dx_ / L / L;
                                vproj[0] += image[ix + cy1 * nx + cz2 * nx * ny] * weight;
                            }
                        }
                    }
                }
            }
            else
            {
                if (-1 <= cy1 && cy1 < ny)
                {
                    if (cz1 == cz2)
                    {
                         if (0 <= cz1 and cz1 < nz)
                         {
                            float ry = (cy2 - yy1) / (yy2 - yy1);
                            if (cy1 >= 0)
                            {
                                float weight = ry * sqrt(1 + ky * ky + kz * kz) * dx_ / L / L;
                                vproj[0] += image[ix + cy1 * nx + cz1 * nx * ny] * weight;
                            }


                            if (cy2 < ny)
                            {
                                float weight = (1 - ry) * sqrt(1 + ky * ky + kz * kz) * dx_ / L / L;
                                vproj[0] += image[ix + cy2 * nx + cz1 * nx * ny] * weight;
                            }
                         }
                    }
                    else if (-1 <= cz1 and cz1 < nz)
                    {
                        float ry = (cy2 - yy1) / (yy2 - yy1);
                        float rz = (cz2 - zz1) / (zz2 - zz1);
                        if (ry > rz)
                        {
                            if (cy1 >= 0 && cz1 >= 0)
                            {
                                float weight = rz * sqrt(1 + ky * ky + kz * kz) * dx_ / L / L;
                                vproj[0] += image[ix + cy1 * nx + cz1 * nx * ny] * weight;

                            }

                            if (cy1 >= 0 && cz2 < nz)
                            {
                                float weight = (ry - rz) * sqrt(1 + ky * ky + kz * kz) * dx_ / L / L;
                                vproj[0] += image[ix + cy1 * nx + cz2 * nx * ny] * weight;
                            }

                            if (cy2 < ny && cz2 < nz)
                            {
                                float weight = (1 - ry) * sqrt(1 + ky * ky + kz * kz) * dx_ / L / L;
                                vproj[0] += image[ix + cy2 * nx + cz2 * nx * ny] * weight;
                            }
                        }
                        else
                        {
                            if (cy1 >= 0 && cz1 >= 0)
                            {
                                float weight = ry * sqrt(1 + ky * ky + kz * kz) * dx_ / L / L;
                                vproj[0] += image[ix + cy1 * nx + cz1 * nx * ny] * weight;
                            }

                            if (cy2 < ny && cz1 >= 0)
                            {
                                float weight = (rz - ry) * sqrt(1 + ky * ky + kz * kz) * dx_ / L / L;
                                vproj[0] += image[ix + cy2 * nx + cz1 * nx * ny] * weight;
                            }

                            if (cy2 < ny && cz2 < nz)
                            {
                                float weight = (1 - rz) * sqrt(1 + ky * ky + kz * kz) * dx_ / L / L;
                                vproj[0] += image[ix + cy2 * nx + cz2 * nx * ny] * weight;
                            }
                        }
                    }
                }

            }
        }
    }
    else
    {
        float kx = xd / yd;
        float kz = zd / yd;

        for (int iy = 0; iy < ny; ++iy)
        {
            float yy1 = iy - ny2;
            float yy2 = yy1 + 1.0f;
            float xx1, xx2, zz1, zz2;

            if (kx >= 0.0f)
            {
                xx1 = (x1 + kx * (yy1 - y1)) + nx2;
                xx2 = (x1 + kx * (yy2 - y1)) + nx2;
            }
            else
            {
                xx1 = (x1 + kx * (yy2 - y1)) + nx2;
                xx2 = (x1 + kx * (yy1 - y1)) + nx2;
            }
            int cx1 = (int)floor(xx1);
            int cx2 = (int)floor(xx2);

            if (kz >= 0.0f)
            {
                zz1 = (z1 + kz * (yy1 - y1)) / dz + nz2;
                zz2 = (z1 + kz * (yy2 - y1)) / dz + nz2;
            }
            else
            {
                zz1 = (z1 + kz * (yy2 - y1)) / dz + nz2;
                zz2 = (z1 + kz * (yy1 - y1)) / dz + nz2;
            }
            int cz1 = (int)floor(zz1);
            int cz2 = (int)floor(zz2);

            if (cx1 == cx2)
            {
                if (0 <= cx1 && cx1 < nx)
                {
                    if (cz1 == cz2)
                    {
                        if (0 <= cz1 && cz1 < nz)
                        {
                            float weight = sqrt(1 + kx * kx + kz * kz) * dx_ / L / L;
                            vproj[0] += image[cx1 + iy * nx + cz1 * nx * ny] * weight;
                        }
                    }
                    else
                    {
                        if (-1 <= cz1 and cz1 < nz)
                        {
                            float rz = (cz2 - zz1) / (zz2 - zz1);
                            if (cz1 >= 0)
                            {
                                float weight = rz * sqrt(1 + kx * kx + kz * kz) * dx_ / L / L;
                                vproj[0] += image[cx1 + iy * nx + cz1 * nx * ny] * weight;
                            }

                            if (cz2 < nz)
                            {
                                float weight = (1 - rz) * sqrt(1 + kx * kx + kz * kz) * dx_ / L / L;
                                vproj[0] += image[cx1 + iy * nx + cz2 * nx * ny] * weight;
                            }
                        }
                    }
                }

            }
            else
            {
                if (-1 <= cx1 && cx1 < nx)
                {
                    if (cz1 == cz2)
                    {
                         if (0 <= cz1 and cz1 < nz)
                         {
                            float rx = (cx2 - xx1) / (xx2 - xx1);
                            if (cx1 >= 0)
                            {
                                float weight = rx * sqrt(1 + kx * kx + kz * kz) * dx_ / L / L;
                                vproj[0] += image[cx1 + iy * nx + cz1 * nx * ny] * weight;
                            }

                            if (cx2 < nx)
                            {
                                float weight = (1 - rx) * sqrt(1 + kx * kx + kz * kz) * dx_ / L / L;
                                vproj[0] += image[cx2 + iy * nx + cz1 * nx * ny] * weight;
                            }
                         }
                    }
                    else if (-1 <= cz1 and cz1 < nz)
                    {
                        float rx = (cx2 - xx1) / (xx2 - xx1);
                        float rz = (cz2 - zz1) / (zz2 - zz1);
                        if (rx > rz)
                        {
                            if (cx1 >= 0 && cz1 >= 0)
                            {
                                float weight = rz * sqrt(1 + kx * kx + kz * kz) * dx_ / L / L;
                                vproj[0] += image[cx1 + iy * nx + cz1 * nx * ny] * weight;
                            }

                            if (cx1 >= 0 && cz2 < nz)
                            {
                                float weight = (rx - rz) * sqrt(1 + kx * kx + kz * kz) * dx_ / L / L /
                                L;
                                vproj[0] += image[cx1 + iy * nx + cz2 * nx * ny] * weight;
                            }

                            if (cx2 < nx && cz2 < nz)
                            {
                                float weight = (1 - rx) * sqrt(1 + kx * kx + kz * kz) * dx_ / L / L;
                                vproj[0] += image[cx2 + iy * nx + cz2 * nx * ny] * weight;
                            }
                        }
                        else
                        {
                            if (cx1 >= 0 && cz1 >= 0)
                            {
                                float weight = rx * sqrt(1 + kx * kx + kz * kz) * dx_ / L / L;
                                vproj[0] += image[cx1 + iy * nx + cz1 * nx * ny] * weight;
                            }

                            if (cx2 < nx && cz1 >= 0)
                            {
                                float weight = (rz - rx) * sqrt(1 + kx * kx + kz * kz) * dx_ / L / L /
                                L;
                                vproj[0]+= image[cx2 + iy * nx + cz1 * nx * ny] * weight;
                            }

                            if (cx2 < nx && cz2 < nz)
                            {
                                float weight = (1 - rz) * sqrt(1 + kx * kx + kz * kz) * dx_ / L / L;
                                vproj[0] += image[cx2 + iy * nx + cz2 * nx * ny] * weight;
                            }
                        }
                    }
                }

            }
        }
    }
//    if (vproj[0] < 0.00000001f) {vproj[0] = 100000000.0f;}

}