#include "hip/hip_runtime.h"
__global__ void kernel_invert(float *mx2, float *my2, hipTextureObject_t tex_mx, hipTextureObject_t tex_my, int nx, int ny);
#include "mex.h"
void mexFunction(int nlhs, mxArray *plhs[], int nrhs, mxArray const *prhs[])
{
// Macro for input and output
#define MX prhs[0]
#define MY prhs[1]
#define PARA prhs[2]

#define OUT_MX plhs[0]
#define OUT_MY plhs[1]

float *h_mx, *h_my;
h_mx = (float*)mxGetData(MX);
h_my = (float*)mxGetData(MY);
int nx, ny;
nx = (int)mxGetScalar(mxGetField(PARA, 0, "nx"));
ny = (int)mxGetScalar(mxGetField(PARA, 0, "ny"));

OUT_MX = mxCreateNumericMatrix(0, 0, mxSINGLE_CLASS, mxREAL);
OUT_MY = mxCreateNumericMatrix(0, 0, mxSINGLE_CLASS, mxREAL);
mwSize outDim[2] = {(mwSize)nx, (mwSize)ny};
mxSetDimensions(OUT_MX, outDim, 2);
mxSetDimensions(OUT_MY, outDim, 2);
mxSetData(OUT_MX, mxMalloc(nx * ny * sizeof(float)));
mxSetData(OUT_MY, mxMalloc(nx * ny * sizeof(float)));


float *h_mx2 = (float*)mxGetData(OUT_MX);
float *h_my2 = (float*)mxGetData(OUT_MY);

hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
hipPitchedPtr dp_mx = make_hipPitchedPtr((void*) h_mx, nx * sizeof(float), nx, ny);
hipPitchedPtr dp_my = make_hipPitchedPtr((void*) h_my, nx * sizeof(float), nx, ny);

hipMemcpy3DParms copyParams = {0};
struct hipExtent extent = make_hipExtent(nx, ny, 1);
copyParams.extent = extent;
copyParams.kind = hipMemcpyHostToDevice;

copyParams.srcPtr = dp_mx;
hipArray *array_mx;
hipMalloc3DArray(&array_mx, &channelDesc, extent);
copyParams.dstArray = array_mx;
hipMemcpy3D(&copyParams);   

copyParams.srcPtr = dp_my;
hipArray *array_my;
hipMalloc3DArray(&array_my, &channelDesc, extent);
copyParams.dstArray = array_my;
hipMemcpy3D(&copyParams);   

hipResourceDesc resDesc;
hipTextureDesc texDesc;
memset(&resDesc, 0, sizeof(resDesc));
resDesc.resType = hipResourceTypeArray;
memset(&texDesc, 0, sizeof(texDesc));
texDesc.addressMode[0] = hipAddressModeClamp;
texDesc.addressMode[1] = hipAddressModeClamp;
texDesc.addressMode[2] = hipAddressModeClamp;
texDesc.filterMode = hipFilterModeLinear;
texDesc.readMode = hipReadModeElementType;
texDesc.normalizedCoords = 0;

resDesc.res.array.array = array_mx;
hipTextureObject_t tex_mx = 0;
hipCreateTextureObject(&tex_mx, &resDesc, &texDesc, NULL);

resDesc.res.array.array = array_my;
hipTextureObject_t tex_my = 0;
hipCreateTextureObject(&tex_my, &resDesc, &texDesc, NULL);

const dim3 gridSize((nx + 16 - 1) / 16, (ny + 16 - 1) / 16, 1);
const dim3 blockSize(16, 16, 4);

float *d_mx2, *d_my2;
hipMalloc((void**)&d_mx2, nx * ny * sizeof(float));
hipMalloc((void**)&d_my2, nx * ny * sizeof(float));

kernel_invert<<<gridSize, blockSize>>>(d_mx2, d_my2, tex_mx, tex_my, nx, ny);
hipDeviceSynchronize();
hipDestroyTextureObject(tex_mx);
hipFreeArray(array_mx);
hipDestroyTextureObject(tex_my);
hipFreeArray(array_my);

hipMemcpy(h_mx2, d_mx2, nx * ny * sizeof(float), hipMemcpyDeviceToHost);
hipMemcpy(h_my2, d_my2, nx * ny * sizeof(float), hipMemcpyDeviceToHost);

hipFree(d_mx2);
hipFree(d_my2);

hipDeviceReset();
return;
}
__global__ void kernel_invert(float *mx2, float *my2, hipTextureObject_t tex_mx, hipTextureObject_t tex_my, int nx, int ny)
{
    int ix = 16 * blockIdx.x + threadIdx.x;
    int iy = 16 * blockIdx.y + threadIdx.y;
    if (ix >= nx || iy >= ny)
        return;
    int id = ix + iy * nx;
    float x = 0, y = 0;
    for (int iter = 0; iter < 10; iter ++){
        x = - tex3D<float>(tex_mx, (x + ix + 0.5f), (y + iy + 0.5f), 0.5f);
        y = - tex3D<float>(tex_my, (x + ix + 0.5f), (y + iy + 0.5f), 0.5f);
    }
    mx2[id] = x;
    my2[id] = y;
}