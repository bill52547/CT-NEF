#include "hip/hip_runtime.h"
#include "cu_add.h"
__host__ void host_add(float *img1, float *img, int nx, int ny, float weight){
    const dim3 gridSize((nx + BLOCKSIZE_X - 1) / BLOCKSIZE_X, (ny + BLOCKSIZE_Y - 1) / BLOCKSIZE_Y, 1);
    const dim3 blockSize(BLOCKSIZE_X, BLOCKSIZE_Y, 1);
    kernel_add<<<gridSize, blockSize>>>(img1, img, nx, ny, weight);
    hipDeviceSynchronize();
}

__global__ void kernel_add(float *img1, float *img, int nx, int ny, float weight){
    int ix = BLOCKSIZE_X * blockIdx.x + threadIdx.x;
    int iy = BLOCKSIZE_Y * blockIdx.y + threadIdx.y;
    
    if (ix >= nx || iy >= ny)
        return;
    int id = ix + iy * nx;
    img1[id] += img[id] * weight;
}

__host__ void host_add2(float *img1, float *img, int nx, int ny, float* img0, float weight, int ind){
    const dim3 gridSize((nx + BLOCKSIZE_X - 1) / BLOCKSIZE_X, (ny + BLOCKSIZE_Y - 1) / BLOCKSIZE_Y, 1);
    const dim3 blockSize(BLOCKSIZE_X, BLOCKSIZE_Y, 1);
    kernel_add2<<<gridSize, blockSize>>>(img1, img, nx, ny, img0, weight, ind);
    hipDeviceSynchronize();
    
}

__global__ void kernel_add2(float *img1, float *img, int nx, int ny, float *img0, float weight, int ind){
    int ix = BLOCKSIZE_X * blockIdx.x + threadIdx.x;
    int iy = BLOCKSIZE_Y * blockIdx.y + threadIdx.y;
    
    if (ix >= nx || iy >= ny)
        return;
    int id = ix + iy * nx;
    float df;
    switch (ind)
    {
        case 1:
            if (ix == nx - 1)
                df = 0.0f;
            else
                df = img0[id + 1] - img0[id];
            break;
        case 2:
            if (iy == ny - 1)
                df = 0.0f;
            else
                df = img0[id + nx] - img0[id];    
            break;
    }
    img1[id] += img[iy + ix * ny] * weight * df;
}