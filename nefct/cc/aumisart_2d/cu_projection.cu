#include "hip/hip_runtime.h"
#include "cu_projection.h"
__host__ void host2_projection(float *proj, float *img, float angle, float SO, float SD, float da, int na, float ai, int nx, int ny)
{
    
}

__host__ void host_projection(float *d_proj, float *d_img, float angle, float SO, float SD, float da, int na, float ai, int nx, int ny)
{
    const dim3 gridSize_singleProj((na + BLOCKSIZE_X - 1) / BLOCKSIZE_X, 1, 1);
    const dim3 blockSize(BLOCKSIZE_X,BLOCKSIZE_Y, 1);
    kernel_projection<<<gridSize_singleProj, blockSize>>>(d_proj, d_img, angle, SO, SD, da, na, ai, nx, ny);
    hipDeviceSynchronize();
}

__global__ void kernel_projection(float *proj, float *img, float angle, float SO, float SD, float da, int na, float ai, int nx, int ny){
    int ia = BLOCKSIZE_X * blockIdx.x + threadIdx.x;
    if (ia >= na)
        return;
    int id = ia;
    proj[id] = 0.0f;
    float x1, y1, x2, y2, x20, y20, cphi, sphi;
    cphi = (float)cosf(angle);
    sphi = (float)sinf(angle);
    x1 = -SO * cphi;
    y1 = -SO * sphi;
    x20 = SD - SO;
    y20 = (ia + ai) * da; // locate the detector cell center before any rotation
    x2 = x20 * cphi - y20 * sphi;
    y2 = x20 * sphi + y20 * cphi;
    float x21, y21; // offset between source and detector center
    x21 = x2 - x1;
    y21 = y2 - y1;

    // y - z plane, where ABS(x21) > ABS(y21)
    if (ABS(x21) > ABS(y21)){
    // if (ABS(cphi) > ABS(sphi)){
        float yi1, yi2;
        int Yi1, Yi2;
        // for each y - z plane, we calculate and add the contribution of related pixels
        for (int ix = 0; ix < nx; ix++){
            // calculate y indices of intersecting voxel candidates
            float xl, xr, yl, yr, ratio;
            float cyll, cylr, cyrl, cyrr, xc;
            xl = x21 - da / 2 * sphi;
            xr = x21 + da / 2 * sphi;
            yl = y21 - da / 2 * cphi;
            yr = y21 + da / 2 * cphi;
            xc = (float)ix + 0.5f - (float)nx / 2 - x1;
            
            ratio = yl / xl;
            cyll = ratio * xc + y1 + ny / 2;
            ratio = yl / xr;
            cylr = ratio * xc + y1 + ny / 2;
            ratio = yr / xl;
            cyrl = ratio * xc + y1 + ny / 2;
            ratio = yr / xr;
            cyrr = ratio * xc + y1 + ny / 2;

            yi1 = MIN4(cyll, cylr, cyrl, cyrr); Yi1 = (int)floorf(yi1);
            yi2 = MAX4(cyll, cylr, cyrl, cyrr); Yi2 = (int)floorf(yi2);

            xc = (float)ix + 0.5f - (float)nx / 2 - x1 ;

            float wy;

            for (int iy = MAX(0, Yi1); iy <= MIN(ny - 1, Yi2); iy++)
            {
                wy = MIN(iy + 1.0f, yi2) - MAX(iy + 0.0f, yi1); wy /= (yi2 - yi1);
                proj[id] += img[ix + iy * nx] * wy / ABS(x21) * sqrt(x21 * x21 + y21 * y21);                
            }        
        }
    }
    // x - z plane, where ABS(x21) <= ABS(y21)    
    else{
        float xi1, xi2;
        int Xi1, Xi2;
        // for each y - z plane, we calculate and add the contribution of related pixels
        for (int iy = 0; iy < ny; iy++){
            // calculate y indices of intersecting voxel candidates
            float yl, yr, xl, xr, ratio;
            float cxll, cxlr, cxrl, cxrr, yc;
            yl = y21 - da / 2 * cphi;
            yr = y21 + da / 2 * cphi;
            xl = x21 - da / 2 * sphi;
            xr = x21 + da / 2 * sphi;
            yc = (float)iy + 0.5f - (float)ny / 2 - y1;
            
            ratio = xl / yl;
            cxll = ratio * yc + x1 + nx / 2;
            ratio = xl / yr;
            cxlr = ratio * yc + x1 + nx / 2;
            ratio = xr / yl;
            cxrl = ratio * yc + x1 + nx / 2;
            ratio = xr / yr;
            cxrr = ratio * yc + x1 + nx / 2;

            xi1 = MIN4(cxll, cxlr, cxrl, cxrr); Xi1 = (int)floorf(xi1);
            xi2 = MAX4(cxll, cxlr, cxrl, cxrr); Xi2 = (int)floorf(xi2);

            yc = (float)iy + 0.5f - (float)ny / 2 - y1;

            float wx;

            for (int ix = MAX(0, Xi1); ix <= MIN(nx - 1, Xi2); ix++)
            {
                wx = MIN(ix + 1.0f, xi2) - MAX(ix + 0.0f, xi1); wx /= (xi2 - xi1);
                proj[id] += img[ix + iy * nx] * wx / ABS(y21) * sqrt(x21 * x21 + y21 * y21);                
            }        
        }            
    }
}