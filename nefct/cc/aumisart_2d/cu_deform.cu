#include "hip/hip_runtime.h"
#include "cu_deform.h"

__host__ void host_deform(float *d_img1, float *d_img, int nx, int ny, float volume, float flow, float *alpha_x, float *alpha_y, float *beta_x, float *beta_y)
{
    const dim3 gridSize((nx + BLOCKSIZE_X - 1) / BLOCKSIZE_X, (ny + BLOCKSIZE_Y - 1) / BLOCKSIZE_Y, 1);
    const dim3 blockSize(BLOCKSIZE_X, BLOCKSIZE_Y, 1);
    float *mx, *my;
    hipMalloc((void**)&mx, nx * ny * sizeof(float));
    hipMalloc((void**)&my, nx * ny * sizeof(float));
    kernel_forwardDVF<<<gridSize, blockSize>>>(mx, my, alpha_x, alpha_y, beta_x, beta_y, volume, flow, nx, ny);
    hipDeviceSynchronize();
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    hipPitchedPtr dp_img = make_hipPitchedPtr((void*) d_img, nx * sizeof(float), nx, ny);
    hipMemcpy3DParms copyParams = {0};
    struct hipExtent extent_img = make_hipExtent(nx, ny, 1);
    copyParams.extent = extent_img;
    copyParams.kind = hipMemcpyDeviceToDevice;
    copyParams.srcPtr = dp_img;
    hipArray *array_img;
    hipMalloc3DArray(&array_img, &channelDesc, extent_img);
    copyParams.dstArray = array_img;
    hipMemcpy3D(&copyParams);   

    hipResourceDesc resDesc;
    hipTextureDesc texDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.addressMode[1] = hipAddressModeClamp;
    texDesc.addressMode[2] = hipAddressModeClamp;
    texDesc.filterMode = hipFilterModeLinear;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = 0;
    resDesc.res.array.array = array_img;
    hipTextureObject_t tex_img = 0;
    hipCreateTextureObject(&tex_img, &resDesc, &texDesc, NULL);
    kernel_deformation<<<gridSize, blockSize>>>(d_img1, tex_img, mx, my, nx, ny);
    hipDeviceSynchronize();
    hipFree(mx);   
    hipFree(my);   
    hipDestroyTextureObject(tex_img);
    hipFreeArray(array_img);

}

__host__ void host_deform_invert(float *d_img1, float *d_img, int nx, int ny, float volume, float flow, float *alpha_x, float *alpha_y, float *beta_x, float *beta_y)
{
    const dim3 gridSize((nx + BLOCKSIZE_X - 1) / BLOCKSIZE_X, (ny + BLOCKSIZE_Y - 1) / BLOCKSIZE_Y, 1);
    const dim3 blockSize(BLOCKSIZE_X, BLOCKSIZE_Y, 1);
    float *mx, *my;
    hipMalloc((void**)&mx, nx * ny * sizeof(float));
    hipMalloc((void**)&my, nx * ny * sizeof(float));
    kernel_forwardDVF<<<gridSize, blockSize>>>(mx, my, alpha_x, alpha_y, beta_x, beta_y, volume, flow, nx, ny);
    hipDeviceSynchronize();

    float *mx2, *my2;
    hipMalloc((void**)&mx2, nx * ny * sizeof(float));
    hipMalloc((void**)&my2, nx * ny * sizeof(float));
    
    host_invert(mx2, my2, mx, my, nx, ny);
    
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    hipPitchedPtr dp_img = make_hipPitchedPtr((void*) d_img, nx * sizeof(float), nx, ny);
    hipMemcpy3DParms copyParams = {0};
    struct hipExtent extent_img = make_hipExtent(nx, ny, 1);
    copyParams.extent = extent_img;
    copyParams.kind = hipMemcpyDeviceToDevice;
    copyParams.srcPtr = dp_img;
    hipArray *array_img;
    hipMalloc3DArray(&array_img, &channelDesc, extent_img);
    copyParams.dstArray = array_img;
    hipMemcpy3D(&copyParams);   

    hipResourceDesc resDesc;
    hipTextureDesc texDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.addressMode[1] = hipAddressModeClamp;
    texDesc.addressMode[2] = hipAddressModeClamp;
    texDesc.filterMode = hipFilterModeLinear;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = 0;
    resDesc.res.array.array = array_img;
    hipTextureObject_t tex_img = 0;
    hipCreateTextureObject(&tex_img, &resDesc, &texDesc, NULL);
    kernel_deformation<<<gridSize, blockSize>>>(d_img1, tex_img, mx2, my2, nx, ny);
    hipDeviceSynchronize();
    hipFree(mx);   
    hipFree(my);   
    
    hipFree(mx2);   
    hipFree(my2);   
    hipDestroyTextureObject(tex_img);
    hipFreeArray(array_img);

}

__host__ void host_invert(float *mx2, float *my2, float *mx, float *my, int nx, int ny)
{
    const dim3 gridSize((nx + BLOCKSIZE_X - 1) / BLOCKSIZE_X, (ny + BLOCKSIZE_Y - 1) / BLOCKSIZE_Y, 1);
    const dim3 blockSize(BLOCKSIZE_X, BLOCKSIZE_Y, 1);
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    hipPitchedPtr dp_mx = make_hipPitchedPtr((void*) mx, nx * sizeof(float), nx, ny);
    hipPitchedPtr dp_my = make_hipPitchedPtr((void*) my, nx * sizeof(float), nx, ny);

    hipMemcpy3DParms copyParams = {0};
    struct hipExtent extent = make_hipExtent(nx, ny, 1);
    copyParams.extent = extent;
    copyParams.kind = hipMemcpyDeviceToDevice;
    
    copyParams.srcPtr = dp_mx;
    hipArray *array_mx;
    hipMalloc3DArray(&array_mx, &channelDesc, extent);
    copyParams.dstArray = array_mx;
    hipMemcpy3D(&copyParams);   

    copyParams.srcPtr = dp_my;
    hipArray *array_my;
    hipMalloc3DArray(&array_my, &channelDesc, extent);
    copyParams.dstArray = array_my;
    hipMemcpy3D(&copyParams);   

    hipResourceDesc resDesc;
    hipTextureDesc texDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.addressMode[1] = hipAddressModeClamp;
    texDesc.addressMode[2] = hipAddressModeClamp;
    texDesc.filterMode = hipFilterModeLinear;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = 0;

    resDesc.res.array.array = array_mx;
    hipTextureObject_t tex_mx = 0;
    hipCreateTextureObject(&tex_mx, &resDesc, &texDesc, NULL);

    resDesc.res.array.array = array_my;
    hipTextureObject_t tex_my = 0;
    hipCreateTextureObject(&tex_my, &resDesc, &texDesc, NULL);

    kernel_invert<<<gridSize, blockSize>>>(mx2, my2, tex_mx, tex_my, nx, ny);
    hipDeviceSynchronize();
    hipDestroyTextureObject(tex_mx);
    hipFreeArray(array_mx);
    hipDestroyTextureObject(tex_my);
    hipFreeArray(array_my);
}
__global__ void kernel_invert(float *mx2, float *my2, hipTextureObject_t tex_mx, hipTextureObject_t tex_my, int nx, int ny)
{
    int ix = BLOCKSIZE_X * blockIdx.x + threadIdx.x;
    int iy = BLOCKSIZE_Y * blockIdx.y + threadIdx.y;
    if (ix >= nx || iy >= ny)
        return;
    int id = ix + iy * nx;
    float x = 0, y = 0;
    for (int iter = 0; iter < 10; iter ++){
        x = - tex3D<float>(tex_mx, (x + ix + 0.5f), (y + iy + 0.5f), 0.5f);
        y = - tex3D<float>(tex_my, (x + ix + 0.5f), (y + iy + 0.5f), 0.5f);
    }
    mx2[id] = x;
    my2[id] = y;
}

__host__ void host_deform2(float *d_img1, float *d_img, int nx, int ny, float volume, float flow, float *alpha_x, float *alpha_y, float *beta_x, float *beta_y)
{
    const dim3 gridSize((nx + BLOCKSIZE_X - 1) / BLOCKSIZE_X, (ny + BLOCKSIZE_Y - 1) / BLOCKSIZE_Y, 1);
    const dim3 blockSize(BLOCKSIZE_X, BLOCKSIZE_Y, 1);
    float *mx, *my;
    hipMalloc((void**)&mx, nx * ny * sizeof(float));
    hipMalloc((void**)&my, nx * ny * sizeof(float));
    kernel_forwardDVF<<<gridSize, blockSize>>>(mx, my, alpha_x, alpha_y, beta_x, beta_y, volume, flow, nx, ny);
    hipDeviceSynchronize();
    kernel_deformation2<<<gridSize, blockSize>>>(d_img1, d_img, mx, my, nx, ny);
    hipDeviceSynchronize();
    hipFree(mx);
    hipFree(my);
}

__global__ void kernel_forwardDVF(float *mx, float *my, float *alpha_x, float *alpha_y, float *beta_x, float *beta_y, float volume, float flow, int nx, int ny)
{
    int ix = BLOCKSIZE_X * blockIdx.x + threadIdx.x;
    int iy = BLOCKSIZE_Y * blockIdx.y + threadIdx.y;
    if (ix >= nx || iy >= ny)
        return;
    int id = ix + iy * nx;    
    mx[id] = alpha_x[id] * volume + beta_x[id] * flow;
    my[id] = alpha_y[id] * volume + beta_y[id] * flow;
}

__global__ void kernel_deformation(float *img1, hipTextureObject_t tex_img, float *mx, float *my, int nx, int ny){
    int ix = BLOCKSIZE_X * blockIdx.x + threadIdx.x;
    int iy = BLOCKSIZE_Y * blockIdx.y + threadIdx.y;
    if (ix >= nx || iy >= ny)
        return;
    int id = ix + iy * nx;
    float xi = ix + mx[id];
    float yi = iy + my[id];
    
    img1[id] = tex3D<float>(tex_img, xi + 0.5f, yi + 0.5f, 0.5f);
}

__global__ void kernel_deformation2(float *img1, float *img, float *mx, float *my, int nx, int ny){
    int ix = BLOCKSIZE_X * blockIdx.x + threadIdx.x;
    int iy = BLOCKSIZE_Y * blockIdx.y + threadIdx.y;
    if (ix >= nx || iy >= ny)
        return;
    int id = iy + ix * ny;
    int id2 = ix + iy * nx;

    float dx, dy;
    if (ix == nx - 1)
        dx = 0;
    else
        dx = img[id2 + 1] - img[id2];
        
    if (iy == ny - 1)
        dy = 0;
    else
        dy = img[id2 + nx] - img[id2];


    img1[id2] = img[id2] + dy * mx[id2] + dx * my[id2];
}