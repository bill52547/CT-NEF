#include "hip/hip_runtime.h"
#include "mex.h"
#define BLOCKSIZE_X 16
#define BLOCKSIZE_Y 16 
#define BLOCKSIZE_Z 1
__host__ void host_diff(float *img1, float *img, int nx, int ny, int ind);
__global__ void kernel_diff(float *img1, float *img, int nx, int ny, int ind);
void mexFunction(int nlhs, mxArray *plhs[], int nrhs, mxArray const *prhs[])
{
// Macro for input and output
#define IN_IMG prhs[0]
#define GEO_PARA prhs[1]
#define OUT_IMG plhs[0]

int nx, ny, ind;

// resolutions of volumes 
if (mxGetField(GEO_PARA, 0, "nx") != NULL)
    nx = (int)mxGetScalar(mxGetField(GEO_PARA, 0, "nx"));
else
	mexErrMsgIdAndTxt("MATLAB:badInput","Can't found valid volume resolution nx.\n");

if (mxGetField(GEO_PARA, 0, "ny") != NULL)
    ny = (int)mxGetScalar(mxGetField(GEO_PARA, 0, "ny"));
else
	mexErrMsgIdAndTxt("MATLAB:badInput","Can't found valid volume resolution ny.\n");

int numImg = nx * ny; // size of image
int numBytesImg = numImg * sizeof(float); // number of bytes in image
if (mxGetField(GEO_PARA, 0, "ind") != NULL)
    ind = (int)mxGetScalar(mxGetField(GEO_PARA, 0, "ind"));
else
	mexErrMsgIdAndTxt("MATLAB:badInput","Can't found ind.\n");
float *h_img;
h_img = (float*)mxGetData(IN_IMG);

float *d_img, *d_img1;
hipMalloc((void**)&d_img, numBytesImg);
hipMalloc((void**)&d_img1, numBytesImg);

hipMemcpy(d_img, h_img, numBytesImg, hipMemcpyHostToDevice);

host_diff(d_img1, d_img, nx, ny, ind);


OUT_IMG = mxCreateNumericMatrix(0, 0, mxSINGLE_CLASS, mxREAL);
const mwSize outDim[2] = {(mwSize)nx, (mwSize)ny};

mxSetDimensions(OUT_IMG, outDim, 2);
mxSetData(OUT_IMG, mxMalloc(numBytesImg));
float *h_outimg = (float*)mxGetData(OUT_IMG);

hipMemcpy(h_outimg, d_img1, numBytesImg, hipMemcpyDeviceToHost);

hipFree(d_img1);
hipFree(d_img);
hipDeviceReset();
return;
}



__host__ void host_diff(float *img1, float *img, int nx, int ny, int ind)
{
    const dim3 gridSize((nx + BLOCKSIZE_X - 1) / BLOCKSIZE_X, (ny + BLOCKSIZE_Y - 1) / BLOCKSIZE_Y);
    const dim3 blockSize(BLOCKSIZE_X, BLOCKSIZE_Y);
    kernel_diff<<<gridSize, blockSize>>>(img1, img, nx, ny, ind);
    hipDeviceSynchronize();
}

__global__ void kernel_diff(float *img1, float *img, int nx, int ny, int ind)
{
    int ix = BLOCKSIZE_X * blockIdx.x + threadIdx.x;
    int iy = BLOCKSIZE_Y * blockIdx.y + threadIdx.y;
    
    if (ix >= nx || iy >= ny)
        return;
    int id = ix + iy * nx;
    switch (ind)
    {    
    case 1:
        if (ix == nx - 1)
            img1[id] = 0.0f;
        else
            img1[id] = img[id + 1] - img[id];
        break;
    case 2:
        if (iy == ny - 1)
            img1[id] = 0.0f;
        else
            img1[id] = img[id + nx] - img[id];
        break;
    }

}